#include "hip/hip_runtime.h"
#include "pop_matrix_cuda_lib.h"
#include <math.h>
#include <stdio.h>


__global__ void pop_matrix_nv(double * G, double * cuboids, 
                              unsigned long long N_cuboids, 
                              unsigned long long Nx, unsigned long long Ny, unsigned long long Npart,
                              double scan_deltax, double scan_deltay,
                              double scan_spacing_x, double scan_spacing_y,
                              double xi0, double eta0, double zeta0, double RefMult,
                              int verbose) {

    double Cm = 1e-7;

    // The thread's unique number 
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_x = gridDim.x * blockDim.x;
    int stride_y = gridDim.y * blockDim.y;

    int threadsInBlock = blockDim.x * blockDim.y;
    int globThread = threadsInBlock * (gridDim.x * blockIdx.y + blockIdx.x)
                     + (blockDim.x * threadIdx.y + threadIdx.x);


    // Loop over sensor measurements. Each sensor is in the xy plane and has area delta^2
    for (unsigned long long sy = idx_y; sy < Ny; sy += stride_y) {
        for (unsigned long long sx = idx_x; sx < Nx; sx += stride_x) {

            // if (sy == idx_y && sx == idx_x) { 
            //     printf("GlobThread : %d sx %ld sy %ld\n", globThread, sx, sy);
            // }

            unsigned long long i_cuboid;
            // unsigned long long i_cuboid_old;
            unsigned long long i_particle_prev;
            unsigned long long i_particle;

            // If grains are not numbered in order this always works
            int i_particle_0_N = 0;

            // Set scan positions in x and y direction
            // unsigned long long i = n % Nx;
            // unsigned long long j = n / Nx;

            // Definitions
            double x, y, z, x2, y2, z2, sign, r2, r, Az, Lx, Ly, F120, F210, F22m;
            double particle_flux[3] = {0};
            double get_flux[3]      = {0};
            double cuboid_center[3] = {0};
            double dr_cuboid[3]     = {0};
            double cuboid_size[3]   = {0};

            double sensor_pos[3] = {0};
            sensor_pos[2] = zeta0;
            sensor_pos[1] = eta0 + scan_spacing_y * sy;
            sensor_pos[0] = xi0 + scan_spacing_x * sx;

            i_cuboid = 0;
            // i_cuboid_old = 0;
            i_particle_prev = (int) cuboids[6];
            i_particle = i_particle_prev;
            i_particle_0_N = 0;

            while (i_cuboid < N_cuboids) {
                if(verbose == 1) {
                    printf("Particle = %lld   Cuboid = %lld i %lld j %lld\n", i_particle, i_cuboid, sx, sy);
                }
                // i_cuboid_old = i_cuboid;

                // The contribution of the flux for mx, my, mz
                for (int k = 0; k < 3; k++) particle_flux[k] = 0.0;

                // Start from the index of the particle being analysed
                // i_particle = (unsigned long long) cuboids[7 * i_cuboid_old + 6];
                // i_cuboid = i_cuboid_old;

                // While the cuboid has particle index of the
                // particle being analysed
                while (i_particle == i_particle_prev) {

                    for (int k = 0; k < 3; k++) {
                        cuboid_center[k] = cuboids[7 * i_cuboid + k];
                        dr_cuboid[k] = sensor_pos[k] - cuboid_center[k];
                        cuboid_size[k] = cuboids[7 * i_cuboid + (k + 3)];
                    }
                    // Cuboid sizes:

                    // calculate flux per cuboid
                    for (int k = 0; k < 3; k++) get_flux[k] = 0.0;

                    for (double s1 = -1; s1 < 1.1;  s1 += 2) {
                        for (double s2 = -1; s2 < 1.1;  s2 += 2) {
                            for (double s3 = -1; s3 < 1.1;  s3 += 2) {
                                for (double s4 = -1; s4 < 1.1;  s4 += 2) {
                                    for (double s5 = -1; s5 < 1.1;  s5 += 2) {
                                        x = RefMult * (dr_cuboid[0] - s1 * cuboid_size[0] + s4 * scan_deltax);
                                        y = RefMult * (dr_cuboid[1] - s2 * cuboid_size[1] + s5 * scan_deltay);
                                        z = RefMult * (dr_cuboid[2] - s3 * cuboid_size[2]);
                                        sign = s1 * s2 * s3 * s4 * s5;
                                        x2 = x * x; y2 = y * y; z2 = z * z;
                                        r2 = x2 + y2 + z2;
                                        r = sqrt(r2);
                                        Az = atan2(x * y, z * r);
                                        if (r != 0.0) {
                                            Lx = log(x + r);
                                            Ly = log(y + r);
                                        } else {
                                            Lx = Ly = 0.0;
                                            printf("Error at p = %lld", i_particle);
                                        }

                                        F120 = 0.5 * ((y2 - z2) * Lx - r * x) - y * (z * Az - x * Ly);
                                        F210 = 0.5 * ((x2 - z2) * Ly - r * y) - x * (z * Az - y * Lx);
                                        F22m = -x * y * Az - z * (x * Lx + y * Ly - r);

                                        get_flux[0] += sign * F120;
                                        get_flux[1] += sign * F210;
                                        get_flux[2] += sign * F22m;
                                    } // s1
                                } // s2
                            } // s3
                        } // s4
                    } // s5

                    // Finish cuboidsloop in the particle i_particle_prev
                    // and continue with the next sensor measurement

                    // scale flux measurement:
                    for (int k = 0; k < 3; k++) particle_flux[k] += -Cm * get_flux[k];
                    i_cuboid += 1;
                    i_particle = (unsigned long long) cuboids[7 * i_cuboid + 6];

                }  // end while cuboids in i_particle

                // printf("Pop: %lld\n", Nx * Ny * (3 * i_particle_0_N    ) + i + Nx * j);
                // printf("Part 0 N: %lld\n", i_particle_0_N);
                // G[sx + sy * Nx][3 * i_particle_0_N    ] = particle_flux[0];
                // G[sx + sy * Nx][3 * i_particle_0_N + 1] = particle_flux[1];
                // G[sx + sy * Nx][3 * i_particle_0_N + 2] = particle_flux[2];

                // Trying to populate G row wise:
                int sensor_idx = Nx * sy + sx;
                G[(3 * Npart) * sensor_idx + (3 * i_particle_0_N    )] = particle_flux[0];
                G[(3 * Npart) * sensor_idx + (3 * i_particle_0_N + 1)] = particle_flux[1];
                G[(3 * Npart) * sensor_idx + (3 * i_particle_0_N + 2)] = particle_flux[2];

                // Move to next particle (last i_particle after ending while loop)
                i_particle_prev = i_particle;
                // Update particle number indexed from 0 to N
                i_particle_0_N += 1;

            } // end while i_cuboids < N_cuboids
        } // end sensor loop sx
    } // end sensor loop sy
}

/*
Main loop to populate the G matrix The outer while loop will last until
reaching the total number of cuboids in the sample. Adjacent cuboids belong to
a single particle, which is indexed in the 6th element of the cuboids array.
The population of the G matrix is performed column wise for every particle. For
each cuboid belonging to a particle, their contribution to the magnetic flux is
summed up for every sensor measurement in steps of delta in the xy plane, which
are given by the loops with the i-j indexes.  The flux is stored column wise.
If Origin is True (default), the cuboids are stored with their original
coordinates. If cuboids are shifted, Origin is False.
*/

// G matrix     -> 1D array that comes from the Python array: (Nx * Ny, 3 * N_parts)
// scan_domain   -> array with 4 entries x1 y1 x2 y2
// cuboids      -> N_cuboids * 7 array
void populate_matrix_cuda(double * G,
                          double * scan_domain, double scan_height,
                          double * cuboids,
                          unsigned long long N_cuboids, unsigned long long Npart,
                          unsigned long long Ny, unsigned long long Nx,
                          double scan_spacing,
                          double scan_deltax, double scan_deltay,
                          int Origin, int verbose
                          ) {


    double RefMult = (scan_height < 0) ? -1.0 : 1.0;
    double xi0, eta0, zeta0;
    if (Origin == 1) {
        xi0 = scan_domain[0];
        eta0 = scan_domain[1];
    } else {
        xi0 = 0.0;
        eta0 = 0.0;
    }
    zeta0 = scan_height;

    size_t G_bytes = sizeof(double) * Nx * Ny * 3 * Npart;
    size_t cuboids_bytes = sizeof(double) * 7 * N_cuboids;

    // Manual mem allocation: G in GPU and cuboids_dev in GPU
    double *G_dev;
    // CUDA_ASSERT(hipMalloc((void**)&G_dev, G_bytes));
    // (allocate in GPU if enough memory, see below)
    // hipMalloc((void**)&G_dev, G_bytes);

    double *cuboids_dev;
    hipMalloc((void**)&cuboids_dev, cuboids_bytes);
    // Copy cuboids array from the host to the GPU
    hipMemcpy(cuboids_dev, cuboids, cuboids_bytes, hipMemcpyHostToDevice);

    // Launch kernel
    // Quadro RTX 6000: 4608 CUDA Cores
    // More refined matrix allocation of blocks if we use smaller n_threads, e.g. 8
    int n_threads = 16; // 256 threads per block in 2D
    // Determine blocks and grid based on problem size: Nx * Ny (easy for a matrix)
    int n_blocks_x = (Nx / n_threads) + ((Nx % n_threads) != 0);
    int n_blocks_y = (Ny / n_threads) + ((Ny % n_threads) != 0);
    dim3 grid(n_blocks_x, n_blocks_y);
    dim3 block(n_threads, n_threads);

    // Checking available memory in GPU:
    size_t free_byte;
    size_t total_byte;
    hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
    double free_db = (double) free_byte / (1024. * 1024.);
    // Quadro RTX 6000: total mem should be 24220.3125 Mb
    double total_db = (double) total_byte / (1024. * 1024.);
    double used_db = total_db - free_db;
    double G_size_mb = (double) G_bytes / (1024. * 1024.);
    double cuboids_size_mb = (double) cuboids_bytes / (1024. * 1024.);

    // if(verbose == 0) {
    printf("------------ Nvidia GPU calculation info ------------\n");
    printf("GPU Memory      (MB): free  = %.4f\n", free_db);
    printf("                      used  = %.4f\n", used_db);
    printf("                      total = %.4f\n", total_db);
    printf("Size of G       (MB): %.4f\n", G_size_mb);
    printf("Size of cuboids (MB): %.4f\n", cuboids_size_mb);
    printf("Blocks grid = %d x %d\n", n_blocks_x, n_blocks_y);
    printf("Threads per block = %d x %d\n", n_threads, n_threads);
    printf("Sensor Matrix dims (rows x cols) = %d x %d\n", Ny, Nx);
    // }

    // Quadro RTX 6000: total mem should be 24220.3125 Mb
    // double MEM_THRESHOLD = 22000;

    // Calculate if there's enough memory in card
    if ((G_size_mb + cuboids_size_mb) > free_db) {

        printf("Not enough available memory in GPU\n");
        // printf("Calculation of large sensor matrices will be implemented in the future\n");
        printf("Stopping calculation\n");
        return;

        // -- WIP --
        // Estimate an optimal size for the sub-matrix. We will use a square m
        // unsigned int N = sqrt((free_db - cuboids_size_mb) / (3 * Npart));
        // G_bytes = sizeof(double) * N * N * 3 * Npart;
        // hipMalloc((void**)&G_dev, G_bytes);

        // TODO:
        // - Use a for loop to populate the submatrix G_dev in device, using strides of
        //   size N in x-direction and y-direction
        //   + For this we have to find out a way to register the i,j sites that were
        //     calculated in the device in order to copy these sites into the
        //     corresponding sites of the full G matrix in the host device
        // - The pop_matrix_nv function in device might need to be modified in order
        //   to specify the i0,j0 pairs where the submatrix is located (starting point
        //   for populating G)
        //   + It might be easy if we just add i0,j0 to the indices sx,sy in the for loop
        //     of pop_matrix_nv
        // - To populate the full G matrix, we can use hipMemcpy with (needs checking)
        //   &G_dev[i0 + N * j0] , to start pop from a different loc in the G array

    }
    else {
        // Allocate G matrix
        hipMalloc((void**)&G_dev, G_bytes);

        // Populate matrix in GPU:
        pop_matrix_nv<<<grid, block>>>(G_dev, cuboids_dev, 
                                       N_cuboids, Nx, Ny, Npart,
                                       scan_deltax, scan_deltay, scan_spacing,
                                       xi0, eta0, zeta0, RefMult, verbose);
        hipDeviceSynchronize();

        // Copy G from the GPU to the host
        hipMemcpy(G, G_dev, G_bytes, hipMemcpyDeviceToHost);
    }

    hipFree(G_dev);
    hipFree(cuboids_dev);

} // main function
